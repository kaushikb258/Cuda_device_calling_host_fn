#include <iostream>
#include "classes.h"
#include "kernel.h"
using namespace std;

#define N 64
#define TPB 16

int main()
{
 cl* cc = new cl[N]; 
 cl* dcc = new cl;
 int size = N*sizeof(cl);

 hipMalloc(&dcc,size);
 hipMemcpy(dcc,cc,size,hipMemcpyHostToDevice);
 fn<<<N/TPB,TPB>>>(dcc,N);
 hipMemcpy(cc,dcc,size,hipMemcpyDeviceToHost);

 for (int i=0; i<N; i++)
 {
  cout<<"i, i-th entry = "<<i<<" "<<cc[i].out()<<endl;
 } 

 hipFree(dcc);   
 delete [] cc; 
}
