#include "hip/hip_runtime.h"
#include "classes.h"
#include "kernel.h"

__global__ void fn(cl* dcc, int n)
{

 const int idx = blockIdx.x*blockDim.x + threadIdx.x;
 dcc[idx].edit(1); 

}
